// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <limits>
#include <assert.h>
#include <stdio.h>

namespace Generators {
namespace cuda {

template <typename T>
__global__ void UpdatePositionIds(T* positions, int batch_beam_size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < batch_beam_size)
    positions[i]++;
}

template <typename T>
__global__ void UpdatePositionIds(T* positions, int total_length, int new_kv_length) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < new_kv_length)
    positions[i] = i + total_length - new_kv_length;
}

template <typename T>
void Launch_UpdatePositionIds(T* positions, int batch_beam_size, int total_length, int new_kv_length, hipStream_t stream) {
  if (batch_beam_size == 1) {
    // For batch size == 1 we calculate position ids with total length and new kv length for continuous decoding
    int threads = std::min(256, new_kv_length);
    int blocks = (new_kv_length + threads - 1) / threads;
    UpdatePositionIds<T><<<blocks, threads, 0, stream>>>(positions, total_length, new_kv_length);
  } else {
    // For batch size > 1 we increment position ids by 1... continuous decoding is not supported
    UpdatePositionIds<T><<<(batch_beam_size + 255) / 256, 256, 0, stream>>>(positions, batch_beam_size);
  }
}

template void Launch_UpdatePositionIds(int32_t* positions, int batch_beam_size, int total_length, int new_kv_length, hipStream_t stream);
template void Launch_UpdatePositionIds(int64_t* positions, int batch_beam_size, int total_length, int new_kv_length, hipStream_t stream);

template <typename T>
__global__ void UpdateAttentionMaskStatic(T* mask_data, int batch_beam_size, int new_kv_length, int total_length, int max_length) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int batch_id = i / new_kv_length;
  int seq_id = (i % new_kv_length) + 1;
  if (i < new_kv_length * batch_beam_size) {
    mask_data[batch_id * max_length + total_length - seq_id] = 1;
  }
}

template <typename T>
__global__ void CopyAndUpdateAttentionMask(T* next_mask_data, const T* mask_data, int batch_beam_size, int new_kv_length, int total_length) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int batch_id = i / total_length;
  int seq_id = i % total_length;
  if (i < total_length * batch_beam_size) {
    if (seq_id < total_length - new_kv_length) {
      next_mask_data[batch_id * total_length + seq_id] = mask_data[batch_id * (total_length - new_kv_length) + seq_id];
    } else {
      next_mask_data[batch_id * total_length + seq_id] = 1;
    }
  }
}

template <typename T>
void Launch_UpdateAttentionMask(T* next_mask_data, T* mask_data, int batch_beam_size, int new_kv_length,
                                int total_length, int max_length, bool update_only, hipStream_t stream) {
  if (update_only) {
    int threads = std::min(256, batch_beam_size * new_kv_length);
    int blocks = (batch_beam_size * new_kv_length + threads - 1) / threads;
    UpdateAttentionMaskStatic<T><<<blocks, threads, 0, stream>>>(mask_data, batch_beam_size, new_kv_length, total_length, max_length);
  } else {
    int threads = std::min(256, batch_beam_size * total_length);
    int blocks = (batch_beam_size * total_length + threads - 1) / threads;
    CopyAndUpdateAttentionMask<T><<<blocks, threads, 0, stream>>>(next_mask_data, mask_data, batch_beam_size, new_kv_length, total_length);
  }
}

template void Launch_UpdateAttentionMask(int32_t* next_mask_data, int32_t* mask_data, int batch_beam_size, int new_kv_length, int total_length, int max_length, bool update_only, hipStream_t stream);
template void Launch_UpdateAttentionMask(int64_t* next_mask_data, int64_t* mask_data, int batch_beam_size, int new_kv_length, int total_length, int max_length, bool update_only, hipStream_t stream);

__global__ void AddLogitsMask(float* batch_logits, int batch_beam_size, int vocab_size, const uint32_t* logits_mask) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= batch_beam_size * vocab_size)
    return;
  int batch_index = index / vocab_size;
  int vocab_index = index % vocab_size;
  if (!(logits_mask[(batch_index * vocab_size + vocab_index) / 32] & (1 << (vocab_index % 32))))
    batch_logits[index] = std::numeric_limits<float>::lowest();
}

void LaunchAddLogitsMask(float* batch_logits, int batch_beam_size, int vocab_size, const uint32_t* logits_mask, hipStream_t stream) {
  int block_size = 256;
  int num_blocks = (batch_beam_size * vocab_size + block_size - 1) / block_size;
  AddLogitsMask<<<num_blocks, block_size, 0, stream>>>(batch_logits, batch_beam_size, vocab_size, logits_mask);
}

__global__ void ConvertFp16ToFp32(const half* src, float* dst, int count) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count)
    dst[idx] = __half2float(src[idx]);
}

void LaunchFp16ToFp32(const uint16_t* fp16, float* fp32, int count, hipStream_t stream) {
  int block_size = 256;
  int num_blocks = (count + block_size - 1) / block_size;
  ConvertFp16ToFp32<<<num_blocks, block_size, 0, stream>>>(reinterpret_cast<const half*>(fp16), fp32, count);
}

__global__ void ConvertFp32ToFp16(const float* src, half* dst, int count) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count)
    dst[idx] = __float2half(src[idx]);
}

void LaunchFp32ToFp16(const float* fp32, uint16_t* fp16, int count, hipStream_t stream) {
  int block_size = 256;
  int num_blocks = (count + block_size - 1) / block_size;
  ConvertFp32ToFp16<<<num_blocks, block_size, 0, stream>>>(fp32, reinterpret_cast<half*>(fp16), count);
}

__global__ void ConvertInt32ToInt64(const int32_t* src, int64_t* dst, int count) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count) {
    dst[idx] = src[idx];
  }
}

void LaunchInt32ToInt64(const int32_t* src, int64_t* dst, int count, hipStream_t stream) {
  int block_size = 256;
  int num_blocks = (count + block_size - 1) / block_size;
  ConvertInt32ToInt64<<<num_blocks, block_size, 0, stream>>>(src, dst, count);
}

namespace {

struct ReorderPastStateParams {
  // Support head_size up to 128
  constexpr static unsigned int kTileSize = 32;
  constexpr static unsigned int kSeqTileSize = 16;
};

}  // namespace

__global__ void ReorderPastStatesKernel(float4* out_buffer,
                                        const float4* in_buffer,
                                        int batch_size,
                                        int num_heads,
                                        int max_length,
                                        int chunked_head_size) {
  __shared__ float4 tile[ReorderPastStateParams::kSeqTileSize][ReorderPastStateParams::kTileSize + 1];

  const int b = blockIdx.z;
  const int n = blockIdx.y;
  const int s_base = blockIdx.x * ReorderPastStateParams::kSeqTileSize;
  const int s = s_base + threadIdx.y;
  const int base_offset = (b * num_heads + n) * max_length * chunked_head_size;

  if (s < max_length) {
    const int in_offset = base_offset + s * chunked_head_size + threadIdx.x;
    tile[threadIdx.y][threadIdx.x] = in_buffer[in_offset];
  }

  __syncthreads();

  const int tidx = threadIdx.x + threadIdx.y * chunked_head_size;
  const int tidx_x = tidx % ReorderPastStateParams::kSeqTileSize;
  const int tidx_y = tidx / ReorderPastStateParams::kSeqTileSize;

  const int s2 = s_base + tidx_x;

  if (s2 < max_length) {
    const int out_offset = base_offset + tidx_y * max_length + s2;
    out_buffer[out_offset] = tile[tidx_x][tidx_y];
  }
}

void ReorderPastStatesKernelLauncher(void* out_buffer,
                                     const void* in_buffer,
                                     int batch_size,
                                     int num_heads,
                                     int max_length,
                                     int head_size,
                                     int chunk_size,
                                     hipStream_t stream) {
  // [B, N, max_length, H2(head_size/chunk_size), equv_chunk_size] -> [B, N, H2(head_size/chunk_size), max_length, equv_chunk_size]
  const int chunked_head_size = head_size / chunk_size;
  const dim3 block(chunked_head_size, ReorderPastStateParams::kSeqTileSize);
  const dim3 grid((max_length + ReorderPastStateParams::kSeqTileSize - 1) / ReorderPastStateParams::kSeqTileSize, num_heads, batch_size);
  if (chunk_size == 4 || chunk_size == 8) {
    ReorderPastStatesKernel<<<grid, block, 0, stream>>>(reinterpret_cast<float4*>(out_buffer),
                                                        reinterpret_cast<const float4*>(in_buffer),
                                                        batch_size,
                                                        num_heads,
                                                        max_length,
                                                        chunked_head_size);
  }
}

__global__ void UpdateCacheIndirectionKernel(int32_t* tgt_indir_cache,
                                             const int32_t* src_indir_cache,
                                             const int32_t* beam_ids,
                                             int batch_size,
                                             int beam_width,
                                             int input_seq_length,
                                             int max_seq_length,
                                             int current_length) {
  int time_step = threadIdx.x + blockIdx.x * blockDim.x;
  int bb_id = threadIdx.y + blockIdx.y * blockDim.y;
  const int batch_id = bb_id / beam_width;
  const int beam_id = bb_id % beam_width;

  if (bb_id >= beam_width * batch_size || time_step >= current_length) {
    return;
  }

  const int src_beam = beam_ids[batch_id * beam_width + beam_id] % beam_width;

  const int tgt_offset = batch_id * beam_width * max_seq_length + beam_id * max_seq_length + time_step;

  if (time_step < input_seq_length) {
    // For time steps that correspond to the input sequence,
    // the beam that it comes from is always 0.
    tgt_indir_cache[tgt_offset] = static_cast<int32_t>(0);
  } else if (time_step == (current_length - 1)) {
    // For the final (newly generated) time step,
    // the beam that it comes from is always the beam that we
    // are currently processing (i.e.) from this point on, these time-steps
    // form the new beams.
    tgt_indir_cache[tgt_offset] = static_cast<int32_t>(beam_id);
  } else {
    // For all other time-steps, we look up the source indirection, to
    // see which beam it came from based on the `src_beam`.
    const int src_offset = batch_id * beam_width * max_seq_length + src_beam * max_seq_length + time_step;
    tgt_indir_cache[tgt_offset] = src_indir_cache[src_offset];
  }
}

void UpdateCacheIndirectionKernelLauncher(int32_t* tgt_indir_cache,
                                          const int32_t* src_indir_cache,
                                          const int32_t* beam_ids,
                                          int batch_size,
                                          int beam_width,
                                          int input_seq_length,
                                          int max_seq_length,
                                          int current_length,
                                          hipStream_t stream) {
  const dim3 block(32);
  const dim3 grid((current_length + block.x - 1) / block.x, batch_size * beam_width);
  UpdateCacheIndirectionKernel<<<grid, block, 0, stream>>>(tgt_indir_cache,
                                                           src_indir_cache,
                                                           beam_ids,
                                                           batch_size,
                                                           beam_width,
                                                           input_seq_length,
                                                           max_seq_length,
                                                           current_length);
}

template <typename T>
__global__ void CopyCrossQKSingleDecodeStepKernel(T* target,  // shape [batch_beam_size, num_alignment_heads, max_length, frames]
                                                  void** qk_layer_pointers,
                                                  int token_index,
                                                  int num_layers,
                                                  int num_heads,
                                                  const int* alignment_heads,
                                                  int frames,
                                                  int max_length,
                                                  int sequence_length) {
  const int pair = blockIdx.x;
  const int num_alignment_heads = gridDim.x;
  const int bbm = blockIdx.y;
  alignment_heads += (pair * 2);
  const int layer = *alignment_heads;
  const int head = *(alignment_heads + 1);

  target += ((int64_t)bbm * num_alignment_heads + pair) * max_length * frames + ((int64_t)token_index * frames);
  T* src = reinterpret_cast<T*>(qk_layer_pointers[layer]) + ((int64_t)bbm * num_heads + head) * sequence_length * frames;

  for (int tid = threadIdx.x; tid < frames; tid += blockDim.x) {
    target[tid] = src[tid];  // use vectorized read write in future if needed
    for (int i = 1; i < sequence_length; i++) {
      target[i * frames + tid] = src[i * frames + tid];
    }
  }
}

template <typename T>
void LaunchCopyCrossQKSingleDecodeStep(hipStream_t stream,
                                       T* cross_qk_buffer_data,
                                       void** qk_layer_pointers,
                                       int token_index,
                                       int batch_beam_size,
                                       int num_layers,
                                       int num_heads,
                                       int num_alignment_heads,
                                       const int* alignment_heads,
                                       int frames,
                                       int max_length,
                                       int sequence_length) {
  dim3 block(512);
  dim3 grid(num_alignment_heads, batch_beam_size);

  if (std::is_same<T, uint16_t>::value) {
    CopyCrossQKSingleDecodeStepKernel<<<grid, block, 0, stream>>>(reinterpret_cast<half*>(cross_qk_buffer_data),
                                                                  qk_layer_pointers,
                                                                  token_index,
                                                                  num_layers,
                                                                  num_heads,
                                                                  alignment_heads,
                                                                  frames,
                                                                  max_length,
                                                                  sequence_length);
  } else {
    CopyCrossQKSingleDecodeStepKernel<<<grid, block, 0, stream>>>(cross_qk_buffer_data,
                                                                  qk_layer_pointers,
                                                                  token_index,
                                                                  num_layers,
                                                                  num_heads,
                                                                  alignment_heads,
                                                                  frames,
                                                                  max_length,
                                                                  sequence_length);
  }
}

template void LaunchCopyCrossQKSingleDecodeStep(hipStream_t stream,
                                                float* cross_qk_buffer_data,
                                                void** qk_layer_pointers,
                                                int token_index,
                                                int batch_beam_size,
                                                int num_layers,
                                                int num_heads,
                                                int num_alignment_heads,
                                                const int* alignment_heads,
                                                int frames,
                                                int max_length,
                                                int sequence_length);

template void LaunchCopyCrossQKSingleDecodeStep(hipStream_t stream,
                                                uint16_t* cross_qk_buffer_data,
                                                void** qk_layer_pointers,
                                                int token_index,
                                                int batch_beam_size,
                                                int num_layers,
                                                int num_heads,
                                                int num_alignment_heads,
                                                const int* alignment_heads,
                                                int frames,
                                                int max_length,
                                                int sequence_length);

template <typename T>
__global__ void CopyDecoderCrossQKAllStepsKernel(int context_decoding_len,
                                                 int num_beams,
                                                 int num_return_sequences,
                                                 int max_length,
                                                 int frames_of_k,
                                                 const T* cross_qk_buffer_data,  // [batch, num_beams, num_alignment_heads, max_length, frames]
                                                 T* cross_qk_output,             // [batch, num_return_sequences, num_alignment_heads, total_decoding_length, frames]
                                                 const int* cache_indir_data) {  // [batch, num_beams, max_length]
  const int pair = blockIdx.y;
  const int num_alignment_heads = gridDim.y;
  const int total_decoding_length = gridDim.x;
  const int token_decoding_index = blockIdx.x;
  const int br = blockIdx.z;
  const int batch = br / num_return_sequences;
  const int ret_seq_id = br % num_return_sequences;

  const int64_t offset_in_cache = ((int64_t)batch * num_return_sequences + ret_seq_id) * max_length + token_decoding_index;
  int bi_src = batch * num_beams + cache_indir_data[offset_in_cache];

  T* target = cross_qk_output + (((int64_t)br * num_alignment_heads + (int64_t)pair) * total_decoding_length + token_decoding_index) * frames_of_k;
  const T* src = cross_qk_buffer_data + (((int64_t)bi_src * num_alignment_heads + (int64_t)pair) * max_length + token_decoding_index) * frames_of_k;
  for (int tid = threadIdx.x; tid < frames_of_k; tid += blockDim.x) {
    target[tid] = src[tid];  // use vectorized read write in future if needed
  }
}

template <typename T>
void LaunchFinalizeCrossQK(hipStream_t stream,
                           int iteration_number,
                           int context_decoding_len,
                           int batch_beam_size,
                           int num_beams,
                           int max_length,
                           int num_alignment_heads,
                           int frames_of_k,
                           const T* cross_qk_buffer_data,
                           T* cross_qk_output,
                           int num_return_sequences,
                           const int* cache_indir_data) {
  int64_t br = (int64_t)batch_beam_size;
  assert(br < 65536L && num_alignment_heads < 65536);

  const int total_decoding_length = iteration_number;
  dim3 block(512);
  dim3 grid(total_decoding_length, num_alignment_heads, (unsigned)br);

  if (std::is_same<T, uint16_t>::value) {
    CopyDecoderCrossQKAllStepsKernel<<<grid, block, 0, stream>>>(context_decoding_len,
                                                                 num_beams,
                                                                 num_return_sequences,
                                                                 max_length,
                                                                 frames_of_k,
                                                                 reinterpret_cast<const half*>(cross_qk_buffer_data),
                                                                 reinterpret_cast<half*>(cross_qk_output),
                                                                 cache_indir_data);
  } else {
    CopyDecoderCrossQKAllStepsKernel<<<grid, block, 0, stream>>>(context_decoding_len,
                                                                 num_beams,
                                                                 num_return_sequences,
                                                                 max_length,
                                                                 frames_of_k,
                                                                 cross_qk_buffer_data,
                                                                 cross_qk_output,
                                                                 cache_indir_data);
  }
}

template void LaunchFinalizeCrossQK(hipStream_t stream,
                                    int iteration_number,
                                    int context_decoding_len,
                                    int batch_beam_size,
                                    int num_beams,
                                    int max_length,
                                    int num_alignment_heads,
                                    int frames_of_k,
                                    const float* cross_qk_buffer_data,
                                    float* cross_qk_output,
                                    int num_return_sequences,
                                    const int* cache_indir_data);

template void LaunchFinalizeCrossQK(hipStream_t stream,
                                    int iteration_number,
                                    int context_decoding_len,
                                    int batch_beam_size,
                                    int num_beams,
                                    int max_length,
                                    int num_alignment_heads,
                                    int frames_of_k,
                                    const uint16_t* cross_qk_buffer_data,
                                    uint16_t* cross_qk_output,
                                    int num_return_sequences,
                                    const int* cache_indir_data);

}  // namespace cuda
}  // namespace Generators
